#include "hip/hip_runtime.h"
/* Dillon Tidgewell
 * ID: 002285452
 * email: tidge101@mail.chapman.edu
 */

#include "book.h"
#include <stdio.h>
#include <cstdlib>
#include <iostream>

using namespace std;

// Kernel function that has each thread save its id
__global__ void save_id(int n, int *dev_tid){
	for(int i = 0; i < n; i++){
		*dev_tid = blockIdx.x;
	}
}

int main(int argc, char* argv[]){
	int numThreads;
	if(argc == 2) {numThreads = atoi(argv[1]);}
	else {numThreads = 512;}
	cout << "Number of threads created: " << numThreads << "\n";

	// Create appropriate variables and allocate memory on device
	int tid;
	int *dev_tid;
	HANDLE_ERROR( hipMalloc( (void**)&dev_tid, sizeof(int) ) );
	// Call kernel function
	save_id<<<numThreads,1>>>(numThreads, dev_tid);

	// Copy thread id's to host
	HANDLE_ERROR( hipMemcpy( &tid, dev_tid, sizeof(int),
														hipMemcpyDeviceToHost ) );

	// Print thread id's
	for(int i = 0; i < numThreads; i++){
		printf("ID of Thread[%d]: %d", i, i.dev_tid);
	}

	// Free the memory we allocated
	hipFree(dev_tid);
}
