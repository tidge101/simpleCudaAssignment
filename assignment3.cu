#include "hip/hip_runtime.h"
/* Dillon Tidgewell
 * ID: 002285452
 * email: tidge101@mail.chapman.edu
 */

#include "book.h"
#include <stdio.h>
#include <cstdlib>
#include <iostream>

using namespace std;

// Kernel function that has each thread save its id
__global__ void save_id(int n, float *tid){
	for(int i = 0; i < n; i++){
		tid[i] = threadIdx.x;
	}
}

int main(int argc, char* argv[]){
	int numThreads;
	if(argc == 2) {numThreads = atoi(argv[1]);}
	else {numThreads = 512;}
	cout << "Number of threads created: " << numThreads << "\n";

	// Create appropriate variables and allocate memory on device
	float *tid, *dev_tid;
	tid = (float*)malloc(numThreads*sizeof(float));
	HANDLE_ERROR( hipMalloc(&dev_tid, numThreads*sizeof(float)));
	// Call kernel function
	save_id<<<1, numThreads>>>(numThreads, dev_tid);

	// Copy thread id's to host
	HANDLE_ERROR( hipMemcpy( tid, dev_tid, numThreads*sizeof(float),
														hipMemcpyDeviceToHost ) );

	// Print thread id's
	for(int i = 0; i < numThreads; i++){
		printf("ID of Thread[%d]: %d\n", i, tid[i]);
	}

	// Free the memory we allocated
	hipFree(dev_tid);
	free(tid);

	return 0;
}
