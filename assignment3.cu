#include "hip/hip_runtime.h"
/* Dillon Tidgewell
 * ID: 002285452
 * email: tidge101@mail.chapman.edu
 */

#include "../common/book.h"
#include <stdio.h>
#include <cstdlib>
#include <iostream>

using namespace std;

// Kernel function that has each thread save its id
__global__ void save_id(int n, ){
	for(int i = 0; i < n; i++){

	}
}

int main(int argc, char* argv[]){
	int numThreads;
	if(argc == 2) {numThreads = atoi(argv[1]);}
	else {numThreads = 512;}
	cout << "Number of threads created: " << numThreads << "\n";

	// Create appropriate variables and allocate memory on device
	int tid;
	int *dev_tid;
	HANDLE_ERROR( hipMalloc( (void**)&dev_tid, sizeof(int) ) );
	// Call kernel function
	save_id<<<numThreads,1>>>;

	// Copy thread id's to host

	// Print thread id's

	// Free the memory we allocated
	hipFree(dev_tid);
}
